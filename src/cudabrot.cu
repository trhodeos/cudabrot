
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SCALE 1/240.0f
#define LIMIT 192
/*
  __global__ void cudabrot_kernel(unsigned char* buffer, unsigned int width, unsigned int height,
  float scale, float limit) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % width;
  int y = index / width;

  if (x < width && y < height) {
  buffer[4 * (y * width + x) + 0] =  0x00;
  buffer[4 * (y * width + x) + 1] =  0x00;
  buffer[4 * (y * width + x) + 2] =  0x00;
  buffer[4 * (y * width + x) + 3] =  0xff;
  }
  }*/

__global__ void cudabrot_kernel(unsigned char* buffer, unsigned int width, unsigned int height,
                                float scale, float limit) {


  __shared__ float sCenter[2];

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % width;
  int y = index / width;

  if (threadIdx.x == 0) {
    sCenter[0] = width / 2.0f;
    sCenter[1] = height / 2.0f;
  }

  __syncthreads();
  if (index < width * height) {
    float ax =  (x - sCenter[0]) * scale;
    float ay =  (y - sCenter[1]) * scale;
    float a1 = ax, b1 = ay;
    float a2, b2;
    int lp = 0;
    while (!(lp > 255 || ((a1*a1 + b1*b1) > limit))) {
      lp++;
      a2 = a1 * a1 - b1 * b1 + ax;
      b2 = 2 * a1 * b1 + ay;
      a1 = a2;
      b1 = b2;
    }
    if (lp > 255) {
      lp = 0;
    }
    __syncthreads();
    // NOTE: OpenGL likes col-major

    buffer[4 * (y * width + x) + 0] = (2*lp) % 256;
    buffer[4 * (y * width + x) + 1] = (3*lp) % 256;
    buffer[4 * (y * width + x) + 2] = (5*lp) % 256;
    buffer[4 * (y * width + x) + 3] = 0xff;
  }
}

void launch_cudabrot_kernel(unsigned char* buffer, unsigned int width, unsigned int height) {
  int num_threads = 256;
  int total_threads = width * height;
  int num_blocks = total_threads / num_threads;
  num_blocks += ((total_threads % num_threads) > 0) ? 1 : 0;
  cudabrot_kernel<<<num_blocks, num_threads>>>(buffer, width, height, SCALE, LIMIT);
  hipError_t e = hipGetLastError();
  if (e != hipSuccess) {
    printf("%s\n", hipGetErrorString(e));
  }
}
