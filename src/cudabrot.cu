
#include <hip/hip_runtime.h>
#include <stdio.h>

#define LIMIT 4

__global__ void cudabrot_kernel(unsigned char* buffer, unsigned int width, unsigned int height,
                                float cx, float cy, float scale) {

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int x = index % width;
  int y = index / width;

  if (index < width * height) {
    float ax = cx + (x - width / 2.0f) * scale;
    float ay = cy + (y - height / 2.0f) * scale;
    float a1 = ax, b1 = ay;
    float a2, b2;
    int lp = 0;

    while (!(lp > 255 || ((a1*a1 + b1*b1) > LIMIT))) {
      lp++;
      a2 = a1 * a1 - b1 * b1 + ax;
      b2 = 2 * a1 * b1 + ay;
      a1 = a2;
      b1 = b2;
    }
    if (lp > 255) {
      lp = 0;
    }
    __syncthreads();
    // NOTE: OpenGL likes col-major

    buffer[4 * (y * width + x) + 0] = (2*lp) % 256;
    buffer[4 * (y * width + x) + 1] = (3*lp) % 256;
    buffer[4 * (y * width + x) + 2] = (5*lp) % 256;
    buffer[4 * (y * width + x) + 3] = 0xff;
  }
}

void launch_cudabrot_kernel(unsigned char* buffer, unsigned int width, unsigned int height,
                            float cx, float cy, float scale) {
  int num_threads = 256;
  int total_threads = width * height;
  int num_blocks = total_threads / num_threads;
  num_blocks += ((total_threads % num_threads) > 0) ? 1 : 0;
  cudabrot_kernel<<<num_blocks, num_threads>>>(buffer, width, height, cx, cy, scale);
  hipError_t e = hipGetLastError();
  if (e != hipSuccess) {
    printf("%s\n", hipGetErrorString(e));
  }
}
